#include "hip/hip_runtime.h"
#include <iostream>
#ifdef USE_NPROF
#include <nvToolsExt.h> 
#endif

#include "parameters.hpp"
#include "knn.cuh"

#ifdef ACC

__device__ static double distance2_cuda(const double3 &a, const double3 &b) {
    double dx = a.x - b.x, dy = a.y - b.y, dz = a.z - b.z;
    return dx*dx + dy*dy + dz*dz;
}

// Morton code or spatial hash for 3D grid cell coordinate
__host__ __device__ static inline unsigned int morton3D(int x, int y, int z) {
    // Interleave bits of x, y, z. Works for up to 10 bits per coordinate.
    unsigned int answer = 0;
    for (unsigned int i = 0; i < 10; ++i) {
        answer |= ((x >> i) & 1) << (3 * i + 0);
        answer |= ((y >> i) & 1) << (3 * i + 1);
        answer |= ((z >> i) & 1) << (3 * i + 2);
    }
    return answer;
}

__device__ static inline int clampi(int v, int a, int b) {
    return v < a ? a : (v > b ? b : v);
}

// Device: locate cell index for a point
__device__ static int get_cell_index(const double3& p, const HashGridParams& params) {
    int cx = int((p.x - params.origin.x) / params.cell_size);
    int cy = int((p.y - params.origin.y) / params.cell_size);
    int cz = int((p.z - params.origin.z) / params.cell_size);
    cx = clampi(cx, 0, params.grid_dim.x - 1);
    cy = clampi(cy, 0, params.grid_dim.y - 1);
    cz = clampi(cz, 0, params.grid_dim.z - 1);
    return (cz * params.grid_dim.y + cy) * params.grid_dim.x + cx;
}

// CUDA kernel using spatial hash grid for KNN
__global__ static void knn_hashgrid_kernel(
    const double3 *points, int numPoints,
    const double3 *queries, int numQueries,
    neighbor *out_results, int k, int nheap, double radius2,
    const HashGrid grid)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= numQueries) return;
    double3 query = queries[tid];
    // Find which cell query is in
    int cx = int((query.x - grid.params.origin.x) / grid.params.cell_size);
    int cy = int((query.y - grid.params.origin.y) / grid.params.cell_size);
    int cz = int((query.z - grid.params.origin.z) / grid.params.cell_size);
    cx = clampi(cx, 0, grid.params.grid_dim.x - 1);
    cy = clampi(cy, 0, grid.params.grid_dim.y - 1);
    cz = clampi(cz, 0, grid.params.grid_dim.z - 1);

    neighbor *results = out_results + tid * k;
    for (int i = 0; i < k; ++i) {
        results[i].idx = -1;
        results[i].dist2 = std::numeric_limits<double>::infinity(); // max distance
    }

    // Neighbor offsets for 3x3x3 and 5x5x5
    const int DX3[27] = {
        0, -1, 0, 1, 0, 0, 0,
        1, 1, 1, 0, 0, 0,
        -1, -1, -1, 1, 1, 1,
        -1, -1, -1, 0, 0, 0, 0, 0
    };
    const int DY3[27] = {
        0, 0, -1, 0, 1, 0, 0,
        1, 0, -1, 1, 0, -1,
        1, 0, -1, 0, 1, -1,
        0, 1, -1, 1, 0, -1, 1, 0
    };
    const int DZ3[27] = {
        0, 0, 0, 0, 0, 1, -1,
        0, 1, -1, 1, -1, 0,
        0, 1, -1, 1, 0, -1,
        1, 0, -1, 1, 1, 1, -1, -1
    };
    // 5x5x5 neighbor offsets: from -2 to +2
    __shared__ int DX5[125], DY5[125], DZ5[125];
    // Only need to initialize once per block
    if (threadIdx.x == 0) {
        int idx = 0;
        for (int dz = -2; dz <= 2; ++dz)
            for (int dy = -2; dy <= 2; ++dy)
                for (int dx = -2; dx <= 2; ++dx) {
                    DX5[idx] = dx;
                    DY5[idx] = dy;
                    DZ5[idx] = dz;
                    idx++;
                }
    }
    __syncthreads();
    // Determine if on boundary
    bool is_on_boundary = (cx == 0 || cy == 0 || cz == 0 ||
                           cx == grid.params.grid_dim.x - 1 ||
                           cy == grid.params.grid_dim.y - 1 ||
                           cz == grid.params.grid_dim.z - 1);
    const int* DX = is_on_boundary ? DX5 : DX3;
    const int* DY = is_on_boundary ? DY5 : DY3;
    const int* DZ = is_on_boundary ? DZ5 : DZ3;
    int range = is_on_boundary ? 125 : 27;

    int n_cand = 0;
    for (int di = 0; di < range; ++di) {
        int ncx = clampi(cx + DX[di], 0, grid.params.grid_dim.x-1);
        int ncy = clampi(cy + DY[di], 0, grid.params.grid_dim.y-1);
        int ncz = clampi(cz + DZ[di], 0, grid.params.grid_dim.z-1);
        int cell_idx = (ncz * grid.params.grid_dim.y + ncy) * grid.params.grid_dim.x + ncx;
        int start = grid.cell_starts[cell_idx];
        int end = grid.cell_starts[cell_idx+1];
        for (int i = start; i < end; ++i) {
            int pi = grid.point_indices[i];
            double d2 = distance2_cuda(points[pi], query);

            if (d2 <= radius2) {
                if (n_cand >= k && results[k - 1].dist2 < d2)
                    continue; // No need to add if we already have k results with smaller distances
                bool is_duplicate = false;
                for (int j = 0; j < k; ++j) {
                    if (results[j].idx == pi) {
                        is_duplicate = true;
                        break;
                    }
                }
                if (is_duplicate) continue;

                for (int ii = 0; ii < k; ++ii) {
                    if (d2 < results[ii].dist2) {
                        // Shift the rest of the result down
                        for (int j = k - 1; j > ii; --j) {
                            results[j] = results[j - 1];
                        }
                        results[ii].idx = pi;
                        results[ii].dist2 = d2;
                        n_cand++;
                        break;
                    }
                }
            }
        }
    }
}

CudaKNN::CudaKNN(const Param& param, const double3_vec& points_vec,
            double resoTimes_) : 
    resolution(param.mesh.resolution),
    points(points_vec.data()), numPoints(points_vec.size()),
    resoTimes(resoTimes_)
{
    build_hash_grid(resolution * resoTimes);

    hipMallocManaged(&d_grid, sizeof(HashGrid));
    *d_grid = grid;

    hipMallocManaged(&d_points, numPoints * sizeof(double3));
    // hipMemcpy(d_points, points, numPoints * sizeof(double3), hipMemcpyHostToDevice);

    // hipDeviceSynchronize();
}

CudaKNN::~CudaKNN()
{
    hipFree(grid.cell_starts);
    hipFree(grid.point_indices);
    hipFree(d_grid);

    hipFree(d_points);
};

// Host: build hash grid (for simplicity, on host then copy to device)
void CudaKNN::build_hash_grid(double cell_size) {
    // Compute bounds
    double3 minp = points[0], maxp = points[0];
    for (int i = 1; i < numPoints; ++i) {
        minp.x = std::min(minp.x, points[i].x);
        minp.y = std::min(minp.y, points[i].y);
        minp.z = std::min(minp.z, points[i].z);
        maxp.x = std::max(maxp.x, points[i].x);
        maxp.y = std::max(maxp.y, points[i].y);
        maxp.z = std::max(maxp.z, points[i].z);
    }
    // Small margin
    double eps = 1e-8;
    minp.x -= eps; minp.y -= eps; minp.z -= eps;
    maxp.x += eps; maxp.y += eps; maxp.z += eps;
    int3 dim;
    dim.x = int((maxp.x - minp.x) / cell_size) + 1;
    dim.y = int((maxp.y - minp.y) / cell_size) + 1;
    dim.z = int((maxp.z - minp.z) / cell_size) + 1;
    int num_cells = dim.x * dim.y * dim.z;
    grid.params.origin = minp;
    grid.params.cell_size = cell_size;
    grid.params.grid_dim = dim;
    grid.num_cells = num_cells;

    // First, count points in each cell
    std::vector<int> cell_counts(num_cells, 0);
    std::vector<unsigned int> cell_codes(numPoints);
    for (int i = 0; i < numPoints; ++i) {
        int cx = int((points[i].x - minp.x) / cell_size);
        int cy = int((points[i].y - minp.y) / cell_size);
        int cz = int((points[i].z - minp.z) / cell_size);
        if (cx < 0) cx = 0; if (cy < 0) cy = 0; if (cz < 0) cz = 0;
        if (cx >= dim.x) cx = dim.x - 1;
        if (cy >= dim.y) cy = dim.y - 1;
        if (cz >= dim.z) cz = dim.z - 1;
        unsigned int code = (cz * dim.y + cy) * dim.x + cx;
        cell_codes[i] = code;
        cell_counts[code]++;
    }
    // Prefix sum for cell_starts
    std::vector<int> cell_starts(num_cells + 1, 0);
    for (int i = 0; i < num_cells; ++i) {
        cell_starts[i + 1] = cell_starts[i] + cell_counts[i];
    }
    // Fill point_indices (bucket sort)
    std::vector<int> next_indices(num_cells, 0);
    std::vector<int> point_indices(numPoints);
    for (int i = 0; i < num_cells; ++i) next_indices[i] = cell_starts[i];
    for (int i = 0; i < numPoints; ++i) {
        int c = cell_codes[i];
        point_indices[next_indices[c]++] = i;
    }
    // Allocate & copy to device
    hipMallocManaged(&grid.cell_starts, sizeof(int) * (num_cells + 1));
    hipMemcpy(grid.cell_starts, cell_starts.data(), sizeof(int) * (num_cells + 1), hipMemcpyHostToDevice);
    hipMallocManaged(&grid.point_indices, sizeof(int) * numPoints);
    hipMemcpy(grid.point_indices, point_indices.data(), sizeof(int) * numPoints, hipMemcpyHostToDevice);
}

void CudaKNN::knnSearchCuda_hashgrid(const double3* queries, int numQueries,
                      neighbor* results, int k, int nheap, 
                      double radius2, double cell_size) {
#ifdef USE_NPROF
    nvtxRangePushA(__FUNCTION__);
#endif

    // points and queries already on device or managed
    // double3* d_points;
    // hipMallocManaged(&d_points, numPoints * sizeof(double3));
    hipMemcpy(d_points, points, numPoints * sizeof(double3), hipMemcpyHostToDevice);

    double3* d_queries;
    hipMallocManaged(&d_queries, numQueries * sizeof(double3));
    hipMemcpy(d_queries, queries, numQueries * sizeof(double3), hipMemcpyHostToDevice);
    neighbor* d_results;
    hipMallocManaged(&d_results, numQueries * k * sizeof(neighbor));

    int threadsPerBlock = 256;
    int numBlocks = (numQueries + threadsPerBlock - 1) / threadsPerBlock;
    knn_hashgrid_kernel<<<numBlocks, threadsPerBlock>>>(
        d_points, numPoints, d_queries, numQueries, d_results, k, nheap, radius2, grid
    );

    cudaCheckSync("knn_hashgrid_kernel");

    hipMemcpy(results, d_results, numQueries * k * sizeof(neighbor), hipMemcpyDeviceToHost);
    // Cleanup
    // hipFree(d_points);
    hipFree(d_queries);
    hipFree(d_results);

#ifdef USE_NPROF
    nvtxRangePop();
#endif
}

void CudaKNN::search_grid(const double3_vec& queries, neighbor_vec& neighbors, 
        int k, double resoTimes)
{
#ifdef USE_NPROF
    nvtxRangePushA(__FUNCTION__);
#endif
    std::cout << "      Running knn query on " << numPoints
            << " points (spatial hash grid)" << std::endl;

    int heapSize = k * 100;
    double maxDist = resoTimes * resolution;
    double cell_size = maxDist;

    long max_size = 1024 * 1024 * 256;
    int nqueries = queries.size();

    int nblocks = (double)nqueries * k / (double)max_size;
    if (nblocks < 1) nblocks = 1;
    printf("        nqueries: %d, k: %d, npoints: %d, max_size: %d, nblocks: %d\n", nqueries, k, numPoints, max_size, nblocks);

    int block_size = (nqueries + nblocks - 1) / nblocks;
    for (int b=0; b<nblocks; b++) {
        int start = b * block_size;
        int end = std::min(start + block_size, nqueries);
        if (start >= end) continue;

        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);
        printf("          Block %3d: %10d to %10d\n", b, start, end);
        printf("            GPU memory: free = %zu MB, total = %zu MB\n", free_mem / (1024 * 1024), total_mem / (1024 * 1024));

        knnSearchCuda_hashgrid(queries.data() + start, end - start,
            neighbors.data() + start*k, k, heapSize, maxDist * maxDist, cell_size);
    }

    hipDeviceSynchronize();
#ifdef USE_NPROF
    nvtxRangePop();
#endif
}
#endif // ACC